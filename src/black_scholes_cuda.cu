#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>
#include <chrono>
#include <fstream>
#include <hip/hip_runtime.h>
#include "utils.h"

__device__ float cnd(float x) {
    return 0.5f * erfcf(-x * M_SQRT1_2);
}

__global__ void blackScholesKernel(float* d_call, float* d_put, float* d_stock, float* d_strike, float* d_time, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        float S = d_stock[i];
        float K = d_strike[i];
        float T = d_time[i];
        float r = 0.05f;
        float sigma = 0.2f;

        float d1 = (logf(S/K) + (r + 0.5f*sigma*sigma)*T) / (sigma*sqrtf(T));
        float d2 = d1 - sigma*sqrtf(T);
        d_call[i] = S * cnd(d1) - K * expf(-r*T) * cnd(d2);
        d_put[i] = K * expf(-r*T) * cnd(-d2) - S * cnd(-d1);
    }
}

void runBlackScholesCUDA() {
    int N = 1 << 20;
    std::vector<float> stock(N, 100.0f), strike(N, 100.0f), time(N, 1.0f);
    std::vector<float> call(N), put(N);

    float *d_stock, *d_strike, *d_time, *d_call, *d_put;
    hipMalloc(&d_stock, N*sizeof(float));
    hipMalloc(&d_strike, N*sizeof(float));
    hipMalloc(&d_time, N*sizeof(float));
    hipMalloc(&d_call, N*sizeof(float));
    hipMalloc(&d_put, N*sizeof(float));

    hipMemcpy(d_stock, stock.data(), N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_strike, strike.data(), N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_time, time.data(), N*sizeof(float), hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();
    blackScholesKernel<<<(N + 255)/256, 256>>>(d_call, d_put, d_stock, d_strike, d_time, N);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    float duration = std::chrono::duration<float, std::milli>(end - start).count();

    hipMemcpy(call.data(), d_call, N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(put.data(), d_put, N*sizeof(float), hipMemcpyDeviceToHost);

    std::ofstream out("data/benchmark_results.csv", std::ios::app);
    out << duration << "\n";
    out.close();

    hipFree(d_stock); hipFree(d_strike); hipFree(d_time);
    hipFree(d_call); hipFree(d_put);
}
